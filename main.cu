#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "matrix-new.h"
#include <math.h>

void blurDemo() {
    int im_size = 31;
    int ker_size = 6;
    int res_size = im_size - ker_size + 1;

    // Initialize some matrices.
    FloatMatrix* image = zeroMatrix(im_size, im_size);
    FloatMatrix* kernal = zeroMatrix(ker_size, ker_size);
    FloatMatrix* result = zeroMatrix(res_size, res_size);
    FloatMatrix* subsampledResult = zeroMatrix(res_size/2, res_size/2);

    // Create example input image. (Operation done on GPU side.)
    drawCircle(image, im_size/2.5, im_size/2.5, im_size/5.0, 0.4);
    drawCircle(image, 1.5*im_size/2.5, 1.5*im_size/2.5, im_size/5.0, 0.1);

    // Create example kernal. (Operation done on GPU side.)
    drawCircle(kernal, ker_size/2 - 0.5, ker_size/2 - 0.5, ker_size/2, 0.1);

    // Apply convolution.
    convolve(image, kernal, result);

    // Subsample by a factor of 2.
    subsample(result, subsampledResult, 2);

    // Wait for GPU.
    hipDeviceSynchronize();

    // Copy data from GPU.
    copyFromDevice(image);
    copyFromDevice(kernal);
    copyFromDevice(result);
    copyFromDevice(subsampledResult);

    // Display matrices.
    displayMatrix(image);
    displayMatrix(kernal);
    displayMatrix(result);
    displayMatrix(subsampledResult);

    // Free allocated memory.
    freeMatrix(image);
    freeMatrix(kernal);
    freeMatrix(result);
}


void sobelDemo() {
    int im_size = 32;
    int ker_size = 3;
    int res_size = im_size - ker_size + 1;

    // Initialize some matrices.
    FloatMatrix* image = zeroMatrix(im_size, im_size);
    FloatMatrix* kernal = zeroMatrix(ker_size, ker_size);
    FloatMatrix* result = zeroMatrix(res_size, res_size);

    // Draw an example input image. This is done on the GPU side.
    drawCircle(image, im_size/3, im_size/3, im_size/3.8, 0.4);
    drawCircle(image, 2*im_size/3, 2*im_size/3, im_size/3.8, 0.1);

    // Create a kernal on the CPU then copy to GPU.
    for (int i=0; i<ker_size; i++) {
        for (int j=0; j<ker_size; j++) {
            kernal->cpu[i*ker_size + j] = (j - ker_size/2.0 + 0.5)/ker_size*2/(1 + abs(i - ker_size/2.0 + 0.5));
        }
    }
    copyToDevice(kernal);
    
    // Apply convolution.
    convolve(image, kernal, result);

    // Wait for GPU.
    hipDeviceSynchronize();
    
    // Copy data from GPU.
    copyFromDevice(image);
    copyFromDevice(kernal);
    copyFromDevice(result);

    // Display matrices.
    displayMatrix(image);
    displaySignedMatrix(kernal); // negative -> red, positive -> blue
    displaySignedMatrix(result);

    // Free allocated memory.
    freeMatrix(image);
    freeMatrix(kernal);
    freeMatrix(result);
}

void kernalReadTest() {
    FloatMatrix** kernals = loadMatrices("../data/conv1-weights.bin", 6, 5, 5);
    FloatMatrix* bias = loadVector("../data/conv1-bias.bin", 6, COLUMN);

    forEach(kernals, 6, displaySignedMatrix);
    displaySignedMatrix(bias);
    
    forEach(kernals, 6, freeMatrix);
    freeMatrix(bias);
}

void imageReadTest() {
    FloatMatrix** numbers = zeroMatrices(10, 28, 28);

    numbers[0] = loadMatrix("../data/0.bin", 28, 28);
    numbers[1] = loadMatrix("../data/1.bin", 28, 28);
    numbers[2] = loadMatrix("../data/2.bin", 28, 28);
    numbers[3] = loadMatrix("../data/3.bin", 28, 28);
    numbers[4] = loadMatrix("../data/4.bin", 28, 28);
    numbers[5] = loadMatrix("../data/5.bin", 28, 28);
    numbers[6] = loadMatrix("../data/6.bin", 28, 28);
    numbers[7] = loadMatrix("../data/7.bin", 28, 28);
    numbers[8] = loadMatrix("../data/8.bin", 28, 28);
    numbers[9] = loadMatrix("../data/9.bin", 28, 28);

    forEach(numbers, 10, displayMatrix);
    forEach(numbers, 10, freeMatrix);
}

void matrixProductTest() {
    FloatMatrix* mat1 = randomMatrix(3, 5);
    FloatMatrix* mat2 = randomMatrix(5, 4);

    FloatMatrix* mat3 = zeroMatrix(3, 4);

    matrixMult(mat1, mat2, mat3);
    copyFromDevice(mat3);

    printMatrix(mat1);
    printMatrix(mat2);
    printMatrix(mat3);
}

int main() {
    srand(time(NULL));

    sobelDemo();

    // blurDemo();

    // kernalReadTest();

    // imageReadTest();

    // matrixProductTest();
    
    return 0;
}
