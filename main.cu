#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "matrix.h"
#include "digit-classifier.h"

void blurDemo() {
    int im_size = 31;
    int ker_size = 6;
    int res_size = im_size - ker_size + 1;

    // Initialize some matrices. (CPU & GPU)
    FloatMatrix* image = zeroMatrix(im_size, im_size);
    FloatMatrix* kernal = zeroMatrix(ker_size, ker_size);
    FloatMatrix* result = zeroMatrix(res_size, res_size);
    FloatMatrix* subsampledResult = zeroMatrix(res_size/2, res_size/2);

    // Create example input image. (GPU)
    drawCircle(image, im_size/2.5, im_size/2.5, im_size/5.0, 0.4);
    drawCircle(image, 1.5*im_size/2.5, 1.5*im_size/2.5, im_size/5.0, 0.1);

    // Create example kernal. (GPU)
    drawCircle(kernal, ker_size/2 - 0.5, ker_size/2 - 0.5, ker_size/2, 0.1);

    // Apply convolution. (GPU)
    convolve(image, kernal, result);

    // Subsample by a factor of 2. (GPU)
    averagePool(result, subsampledResult, 2);

    // Wait for GPU.
    hipDeviceSynchronize();

    // Copy data from GPU to CPU.
    copyFromDevice(image);
    copyFromDevice(kernal);
    copyFromDevice(result);
    copyFromDevice(subsampledResult);

    // Display matrices. (CPU)
    displayMatrix(image);
    displayMatrix(kernal);
    displayMatrix(result);
    displayMatrix(subsampledResult);

    // Free allocated memory.
    freeMatrix(image);
    freeMatrix(kernal);
    freeMatrix(result);
}


void sobelDemo() {
    int im_size = 32;
    int ker_size = 3;
    int res_size = im_size - ker_size + 1;

    // Initialize some matrices.
    FloatMatrix* image = zeroMatrix(im_size, im_size);
    FloatMatrix* kernal = zeroMatrix(ker_size, ker_size);
    FloatMatrix* result = zeroMatrix(res_size, res_size);

    // Draw an example input image. (GPU)
    drawCircle(image, im_size/3, im_size/3, im_size/3.8, 0.4);
    drawCircle(image, 2*im_size/3, 2*im_size/3, im_size/3.8, 0.1);

    // Create a kernal on the CPU then copy to GPU.
    for (int i=0; i<ker_size; i++) {
        for (int j=0; j<ker_size; j++) {
            kernal->cpu[i*ker_size + j] = (j - ker_size/2.0 + 0.5)/ker_size*2/(1 + abs(i - ker_size/2.0 + 0.5));
        }
    }
    copyToDevice(kernal);
    
    // Apply convolution. (GPU)
    convolve(image, kernal, result);

    // Wait for GPU.
    hipDeviceSynchronize();
    
    // Copy data from GPU to CPU.
    copyFromDevice(image);
    copyFromDevice(kernal);
    copyFromDevice(result);

    // Display matrices. (CPU)
    displayMatrix(image);
    displaySignedMatrix(kernal); // negative -> red, positive -> blue
    displaySignedMatrix(result);

    // Free allocated memory.
    freeMatrix(image);
    freeMatrix(kernal);
    freeMatrix(result);
}

void paramsReadTest() {
    // Read matrices from files.
    FloatMatrix** kernals = loadMatrices("data/conv1-weights.bin", 6, 5, 5);
    FloatMatrix* bias = loadVector("data/conv1-bias.bin", 6, COLUMN);

    // Display them.
    forEachMatrix(kernals, 6, displaySignedMatrix);
    displaySignedMatrix(bias);
    
    // Free allocated memory.
    forEachMatrix(kernals, 6, freeMatrix);
    freeMatrix(bias);
}

void imageReadTest() {
    // Initialize 10 28x28 matrices to store images.
    FloatMatrix** numbers = zeroMatrices(10, 28, 28);

    // Load all 10 images.
    numbers[0] = loadMatrix("data/0.bin", 28, 28);
    numbers[1] = loadMatrix("data/1.bin", 28, 28);
    numbers[2] = loadMatrix("data/2.bin", 28, 28);
    numbers[3] = loadMatrix("data/3.bin", 28, 28);
    numbers[4] = loadMatrix("data/4.bin", 28, 28);
    numbers[5] = loadMatrix("data/5.bin", 28, 28);
    numbers[6] = loadMatrix("data/6.bin", 28, 28);
    numbers[7] = loadMatrix("data/7.bin", 28, 28);
    numbers[8] = loadMatrix("data/8.bin", 28, 28);
    numbers[9] = loadMatrix("data/9.bin", 28, 28);

    // Display them as ASCII art.
    forEachMatrix(numbers, 10, displayMatrix);

    // Fre allocated memory.
    forEachMatrix(numbers, 10, freeMatrix);
}

void matrixMultiplicationTest() {
    srand(time(NULL));

    // Choose random matrix sizes.
    int m = 2 + rand() % 6;
    int n = 2 + rand() % 6;
    int p = 2 + rand() % 6;

    // Initialize some random matrices.
    FloatMatrix* mat1 = randomMatrix(m, n);
    FloatMatrix* mat2 = randomMatrix(n, p);

    // Initialize a matrix to store the result of the matrix multiplication.
    FloatMatrix* result = zeroMatrix(m, p);

    // Multiply matrix on the GPU and copy result to CPU.
    matrixMult(mat1, mat2, result);
    hipDeviceSynchronize();
    copyFromDevice(result);

    // Print all the matrices.
    printMatrix(mat1);
    printMatrix(mat2);
    printMatrix(result);

    // Free allocated memory.
    freeMatrix(mat1);
    freeMatrix(mat2);
    freeMatrix(result);
}

void digitClassifierDemo() {
    ConvolutionLayer* conv1 = newConvolutionLayer(1, 6, 5, 5, 28, 28, TANH);
    loadConvolutionLayerParams(conv1, "data/conv1-weights.bin", "data/conv1-bias.bin");

    ConvolutionLayer* conv2 = newConvolutionLayer(6, 16, 5, 5, 12, 12, TANH);
    loadConvolutionLayerParams(conv2, "data/conv2-weights.bin", "data/conv2-bias.bin");

    DenseLayer* dense1 = newDenseLayer(16 * 4 * 4, 120, TANH);
    loadDenseLayerParams(dense1, "data/dense1-weights.bin", "data/dense1-bias.bin");

    DenseLayer* dense2 = newDenseLayer(120, 84, TANH);
    loadDenseLayerParams(dense2, "data/dense2-weights.bin", "data/dense2-bias.bin");

    DenseLayer* dense3 = newDenseLayer(84, 10, SOFTMAX);
    loadDenseLayerParams(dense3, "data/dense3-weights.bin", "data/dense3-bias.bin");

    NeuralNetwork* cnn = newNeuralNetwork();

    addLayer(cnn, (Layer*) conv1);
    addLayer(cnn, (Layer*) newAveragePoolingLayer(6, 24, 24, 2));
    addLayer(cnn, (Layer*) conv2);
    addLayer(cnn, (Layer*) newAveragePoolingLayer(16, 8, 8, 2));
    addLayer(cnn, (Layer*) newFlattenLayer(16, 4, 4));
    addLayer(cnn, (Layer*) dense1);
    addLayer(cnn, (Layer*) dense2);
    addLayer(cnn, (Layer*) dense3);

    FloatMatrix* imageOfNumber = loadMatrix("data/3.bin", 28, 28);
    FloatMatrix** input = &imageOfNumber;

    displayMatrix(imageOfNumber);

    FloatMatrix** output = forward(cnn, input);
    
    printf("\nConv 1 outputs:\n");
    displayConvolutionLayerOutputs(conv1);

    printf("\nConv 2 outputs:\n");
    displayConvolutionLayerOutputs(conv2);

    printf("\nDense 3 output:\n");
    copyFromDevice(dense3->output[0]);
    printMatrix(dense3->output[0]);
}

int main() {
    // blurDemo();
    // sobelDemo();
    // paramsReadTest();
    // imageReadTest();
    // matrixMultiplicationTest();

    digitClassifierDemo();
    
    return 0;
}