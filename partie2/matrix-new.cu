#include "hip/hip_runtime.h"
#include "matrix-new.h"

__host__ FloatMatrix* newMatrix(float* cpu, int m, int n) {
    FloatMatrix* matrix = (FloatMatrix*) malloc(sizeof(FloatMatrix));
    matrix->cpu = cpu;
    matrix->m = m;
    matrix->n = n;
    hipMalloc(&(matrix->gpu), sizeof(float) * m * n);
    copyToDevice(matrix);
    return matrix;
}

__host__ void freeMatrix(FloatMatrix* matrix) {
    free(matrix->cpu);
    hipFree(matrix->gpu);
    free(matrix);
}

__host__ FloatMatrix* zeroMatrix(int m, int n) {
    FloatMatrix* matrix = newMatrix(
        (float*) calloc(sizeof(float), m * n),
        m,
        n
    );
    copyToDevice(matrix);
    return matrix;
}

__host__ FloatMatrix** zeroMatrices(int count, int m, int n) {
    FloatMatrix** matrices = (FloatMatrix**) malloc(count * sizeof(FloatMatrix*));
    for (int i=0; i<count; i++) {
        matrices[i] = zeroMatrix(m, n);
    }
    return matrices;
}

__host__ FloatMatrix* randomMatrix(int n, int m) {
    float* cpuMatrix = (float*) malloc(sizeof(float) * m * n);
    for (int i=0; i<m*n; i++) {
        cpuMatrix[i] = (rand() % RAND_MAX) / (float) RAND_MAX;
    }
    return newMatrix(cpuMatrix, n, m);
}

__host__ FloatMatrix** randomMatrices(int count, int m, int n) {
    FloatMatrix** matrices = (FloatMatrix**) malloc(count * sizeof(FloatMatrix*));
    for (int i=0; i<count; i++) {
        matrices[i] = randomMatrix(m, n);
    }
    return matrices;
}

__host__ void forEach(FloatMatrix** matrices, int count, void (*fun)(FloatMatrix* matrix)) {
    for (int i=0; i<count; i++) {
        fun(matrices[i]);
    }
}

__host__ void copyToDevice(FloatMatrix* matrix) {
    hipMemcpy(
        matrix->gpu, 
        matrix->cpu, 
        sizeof(float)*matrix->m*matrix->n, 
        hipMemcpyHostToDevice
    );
}

__host__ void copyFromDevice(FloatMatrix* matrix) {
    hipMemcpy(
        matrix->cpu, 
        matrix->gpu, 
        sizeof(float)*matrix->m*matrix->n, 
        hipMemcpyDeviceToHost
    );
}

__host__ void printMatrix(FloatMatrix* matrix) {
    int l = matrix->n * matrix->m;
    printf("Matrix([\n");
    for (int i=0; i<l; i++) {
        if (i % matrix->m == 0) {
            printf("  [ %4.1f,", matrix->cpu[i]);
        } else if (i % matrix->m == matrix->m-1) {
            printf(" %4.1f ],\n", matrix->cpu[i]);
        } else {
            printf(" %4.1f,", matrix->cpu[i]);
        }
    }
    printf("])\n");
}

__host__ void displayMatrix(FloatMatrix* matrix) {
    char levels[] = " .:;+=xX$&";
    int l = matrix->n * matrix->m;
    printf("@@@@");
    for (int i=0; i<matrix->m+2; i++)
        printf("@@");
    printf("\n@@");
    for (int i=0; i<matrix->m+2; i++)
        printf("  ");
    printf("@@\n@@  ");
    for (int i=0; i<l; i++) {
        float val = matrix->cpu[i];
        int lev = (int) (val * 10);
        if (lev > 9) lev = 9;
        if (lev < 0) lev = 0;
        printf("%c%c", levels[lev], levels[lev]);
        if (i % matrix->m == matrix->m-1) {
            printf("  @@\n@@  ");
        }
    }

    for (int i=0; i<matrix->m+1; i++)
        printf("  ");
    printf("@@\n@@");
    for (int i=0; i<matrix->m+2; i++)
        printf("@@");
    printf("@@\n");
}

__host__ void displaySignedMatrix(FloatMatrix* matrix) {
    char levels[] = " .:;+=xX$&";
    int l = matrix->n * matrix->m;
    printf("%s@@@@", RESET);
    for (int i=0; i<matrix->m+2; i++)
        printf("@@");
    printf("\n@@");
    for (int i=0; i<matrix->m+2; i++)
        printf("  ");
    printf("%s@@\n@@  ", RESET);
    for (int i=0; i<l; i++) {
        float val = matrix->cpu[i];

        int lev = (int) (abs(val) * 10);
        if (lev > 9) lev = 9;
        if (lev < 0) lev = 0;
        if (val > 0)
            printf("%s%c%c", KBLU, levels[lev], levels[lev]);
        else 
            printf("%s%c%c", KRED, levels[lev], levels[lev]);
        if (i % matrix->m == matrix->m-1) {
            printf("%s  @@\n@@  ", RESET);
        }
    }

    for (int i=0; i<matrix->m+1; i++)
        printf("  ");
    printf("%s@@\n@@", RESET);
    for (int i=0; i<matrix->m+2; i++)
        printf("@@");
    printf("@@\n");
}

__global__ void convolveGpu(float* image, float* kernal, float* result, int im_m, int im_n, int ker_m, int ker_n) {
    int res_i = threadIdx.x;
    int res_j = blockIdx.x;
    int ker_i, ker_j;
    int im_i, im_j;
    float sum = 0;
    for (int i=0; i<ker_m*ker_n; i++) {
        ker_i = i / ker_n;
        ker_j = i % ker_n;
        im_i = res_i + ker_i;
        im_j = res_j + ker_j;
        sum += image[im_i*im_n + im_j] * kernal[ker_i*ker_n + ker_j];
    }
    result[res_i*blockDim.x + res_j] = sum;
}

__host__ void convolve(FloatMatrix* image, FloatMatrix* kernal, FloatMatrix* result) {
    copyToDevice(image);
    copyToDevice(kernal);
    convolveGpu<<<image->m - kernal->m + 1, image->n - kernal->n + 1>>>(
        image->gpu, kernal->gpu, result->gpu, image->m, image->n, kernal->m, kernal->n
    );
    copyFromDevice(result);
}

__host__ void drawCircle(FloatMatrix* matrix, float x, float y, float r, float color) {
    for (int i=0; i<matrix->m; i++) {
        for (int j=0; j<matrix->n; j++) {
            if (sqrt((i-y)*(i-y) + (j-x)*(j-x)) < r) {
                matrix->cpu[i*matrix->n + j] = color;
            }
        }
    }
    copyToDevice(matrix);
}